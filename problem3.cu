#include <stdio.h>
#include <math.h>
#include <hip/hip_runtime.h>

#include <iostream>
#include <vector>
#include <chrono>

using namespace std::chrono;

#define BLOCKSIZE 64

__global__ void matmul1(int N, const float *A, const float *B, float *C) {

  const int i = blockIdx.x * blockDim.x + threadIdx.x;
  const int j = blockIdx.y * blockDim.y + threadIdx.y;

  if (i < N && j < N) {
    float val = 0.f;
    for (int k = 0; k < N; ++k) {
      val += A[k + i * N] * B[j + k * N];
    }
    C[j + i * N] += val;    
  }
}

__global__ void matmul2(int N, const float *A, const float *B, float *C) {

  const int j = blockIdx.x * blockDim.x + threadIdx.x;
  const int i = blockIdx.y * blockDim.y + threadIdx.y;

  //  const int i = blockIdx.x * BLOCKSIZE + (threadIdx.x / BLOCKSIZE);
  //  const int j = blockIdx.y * BLOCKSIZE + (threadIdx.x % BLOCKSIZE);  

  if (i < N && j < N) {
    float val = 0.f;
    for (int k = 0; k < N; ++k) {
      val += A[k + i * N] * B[j + k * N];
    }
    C[j + i * N] += val;    
  }
}

__global__ void matmul3(int N, const float *A, const float *B, float *C) {
  
  // the output block that we want to compute in this threadblock
  const int cRow = blockIdx.x;
  const int cCol = blockIdx.y;

  // allocate buffer for current block in fast shared mem
  // shared mem is shared between all threads in a block
  __shared__ float s_A[BLOCKSIZE * BLOCKSIZE];
  __shared__ float s_B[BLOCKSIZE * BLOCKSIZE];

  // the inner row & col that we're accessing in this thread
  const int threadCol = threadIdx.x % BLOCKSIZE;
  const int threadRow = threadIdx.x / BLOCKSIZE;

  // advance pointers to the starting positions
  A += cRow * BLOCKSIZE * N;                    // row=cRow, col=0
  B += cCol * BLOCKSIZE;                        // row=0, col=cCol
  C += cRow * BLOCKSIZE * N + cCol * BLOCKSIZE; // row=cRow, col=cCol

  float tmp = 0.0;
  for (int blkIdx = 0; blkIdx < N; blkIdx += BLOCKSIZE) {
    // Have each thread load one of the elements in A & B
    // Make the threadCol (=threadIdx.x) the consecutive index
    // to allow global memory access coalescing
    s_A[threadRow * BLOCKSIZE + threadCol] = A[threadRow * N + threadCol];
    s_B[threadRow * BLOCKSIZE + threadCol] = B[threadRow * N + threadCol];

    // block threads in this block until cache is fully populated
    __syncthreads();

    A += BLOCKSIZE;
    B += BLOCKSIZE * N;

    // execute the dotproduct on the currently cached block
    for (int dotIdx = 0; dotIdx < BLOCKSIZE; ++dotIdx) {
      tmp += s_A[threadRow * BLOCKSIZE + dotIdx] * s_B[dotIdx * BLOCKSIZE + threadCol];
    }
    // need to sync again at the end, to avoid faster threads
    // fetching the next block into the cache before slower threads are done
    __syncthreads();
  }
  C[threadRow * N + threadCol] += tmp;
}

int main(int argc, char * argv[]){


for (int version = 1; version <=3; ++version){
    // print out the version 'version'

    // print out the version number
    printf("Version %d", version);
    printf("\n");

    
    for (int power = 9; power <= 12; ++power){
    int N = pow(2,power);
    // print out N
    printf("N = %d", N);
    printf("\n");

    
  if (argc > 1){
    N = atoi(argv[1]);
  }

  float * A = new float[N * N];
  float * B = new float[N * N];
  float * C = new float[N * N];

  for (int i = 0; i < N * N; ++i){
    A[i] = 0.f;
    B[i] = 0.f;
    C[i] = 0.f;
  }
  for (int i = 0; i < N; ++i){
    A[i + i * N] = 1.f; // identity
    B[i + i * N] = 1.f; // identity
  }

  // allocate memory and copy to the GPU
  float * d_A;
  float * d_B;
  float * d_C;
  int size = N * N * sizeof(float);
  hipMalloc((void **) &d_A, size);
  hipMalloc((void **) &d_B, size);
  hipMalloc((void **) &d_C, size);
  
  // copy memory over to the GPU
  hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
  hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);
  hipMemcpy(d_C, C, size, hipMemcpyHostToDevice);

  // Next largest multiple of blockSize
  int numBlocks = (N + BLOCKSIZE - 1) / BLOCKSIZE; 
//   printf("N = %d, numBlocks * blockSize = %d\n", N, numBlocks * BLOCKSIZE);
  dim3 gridDims(numBlocks, numBlocks);
  dim3 blockDims(BLOCKSIZE, BLOCKSIZE);

// int num_trials = 10;

int num_trials = 1;
if (version ==1) {

    #if 1
    float time;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    for (int i = 0; i < num_trials; ++i){
        matmul1 <<< gridDims, blockDims >>> (N, d_A, d_B, d_C);
    }

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);
    
    printf("Time to run kernel 10x: %6.2f ms.\n", time);
    #endif

}

if (version ==2) {
    #if 1
    float time2;
    hipEvent_t start2, stop2;
    hipEventCreate(&start2);
    hipEventCreate(&stop2);
    hipEventRecord(start2, 0);

    for (int i = 0; i < num_trials; ++i){
        matmul2 <<< gridDims, blockDims >>> (N, d_A, d_B, d_C);
    }

    hipEventRecord(stop2, 0);
    hipEventSynchronize(stop2);
    hipEventElapsedTime(&time2, start2, stop2);
    
    printf("Time to run kernel 10x: %6.2f ms.\n", time2);
    #endif
}

if (version ==3) {
    #if 1
    float time3;
    hipEvent_t start3, stop3;
    hipEventCreate(&start3);
    hipEventCreate(&stop3);
    hipEventRecord(start3, 0);

    for (int i = 0; i < num_trials; ++i){
        matmul3 <<< gridDims, blockDims >>> (N, d_A, d_B, d_C);
    }

    hipEventRecord(stop3, 0);
    hipEventSynchronize(stop3);
    hipEventElapsedTime(&time3, start3, stop3);
    
    printf("Time to run kernel 10x: %6.2f ms.\n", time3);
    #endif
}
    
  


}
}

return 0;
}
    



  

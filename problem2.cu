
#include <stdio.h>
#include <math.h>
#include <hip/hip_runtime.h>


#include <iostream>
#include <vector>
#include <chrono>

using namespace std::chrono;


__global__ void add(int N, const float *x, float *y, int blocksize){

   extern __shared__ float s_x[]; // shared memory for x


  const int i = blockDim.x * blockIdx.x + threadIdx.x;
  const int tid = threadIdx.x;
  
  // coalesced reads in
  s_x[tid] = 0.f;
  
  
  if (i < N){
    if (tid <blockDim.x + 2) {
        if(i == 0) {
            s_x[tid] = x[0];
        }
        else if(i == N-1) {
            s_x[tid] = x[N-1];
        }
        else {
            s_x[tid] = x[i-1];
        }
        
    
  }

  // number of "live" threads per block
  
  __syncthreads(); 
  
                                             // I add +1 to the index so it adjusts for the shared memory, which has been shifted 1 unit
  if (i<N){

  y[i]= -s_x[tid + 1+1] + 2* s_x[tid+1] - s_x[tid-1+1];

  }

}
}


__global__ void add2(int N, const float *x, float *y, int blocksize){
  
  int i = blockIdx.x * blockDim.x + threadIdx.x;  
  if (i < N-1 && i > 0){
    y[i] = -x[i+1] +2*x[i] - x[i-1];
  }

  if (i==0){
    y[i] = -x[i+1] +2*x[i] - x[i];
  }

  if (i==N-1){
    y[i] = -x[i] +2*x[i] - x[i-1];
  }


  if (i>0 && i <N-1){
    y[i] = -x[i+1] +2*x[i] - x[i-1];
}
}

void printArray(const float* y, int N) {
    std::cout << "[ ";
    for (int i = 0; i < 100; ++i) {
        std::cout << y[i] << " ";
    }
    std::cout << " ]" << std::endl;
}

int main(void){

  int N = 1e6;
  float * x = new float[N];
  float * y = new float[N];

  for (int i = 0; i < N; ++i){
    x[i] = 1.f;
  }

  int size = N * sizeof(float);

  // allocate memory and copy to the GPU
  float * d_x;
  float * d_y;
  hipMalloc((void **) &d_x, size);
  hipMalloc((void **) &d_y, size);
  
  // copy memory over to the GPU
  hipMemcpy(d_x, x, size, hipMemcpyHostToDevice);
  hipMemcpy(d_y, y, size, hipMemcpyHostToDevice);

  // call the add function  

                                                            //return here  

    int blockSize = 65536;
    
  
  int numBlocks = (N + blockSize - 1) / blockSize;
                                                                              // TIMER
  

    #if 1

    float time;                                                                          
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);


    for (int trials = 0; trials < 10; ++trials){
    add2<<<numBlocks, blockSize>>>(N, d_x, d_y,blockSize);

    }
  
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);
   
    printf("Time to run kernel 10x: %6.3f ms.\n", time);
    #endif
// 
    hipMemcpy(y, d_y, size, hipMemcpyDeviceToHost);
                                                                                // Verify check
    for (int i = 0; i < N; ++i){
    if(y[i] != 0) {
        std::cout << "ERROR: Non Zero!";
    }
    }
    std::cout << "\n";
    
   
                                 // SECOND VERSION   
#if 1

float time2;                                                                          
hipEvent_t start2, stop2;
hipEventCreate(&start2);
hipEventCreate(&stop2);
hipEventRecord(start2, 0);



for (int trials = 0; trials < 10; ++trials){
  add<<<numBlocks, blockSize>>>(N, d_x, d_y,blockSize);
  }    
  
    hipEventRecord(stop2, 0);
    hipEventSynchronize(stop2);
    hipEventElapsedTime(&time2, start2, stop2);
   
    printf("Time to run kernel 10x: %6.3f ms.\n", time2);

  // copy memory back to the CPU
#endif



return 0;
}


